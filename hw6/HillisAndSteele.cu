#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "config.h"

void fill_array(DTYPE *a, size_t n);
void prefix_sum(DTYPE *a, size_t n);
void print_array(DTYPE *a, size_t n);
void ensure(hipError_t error, const char *msg);

int main(int argc, char *argv[]) {
	// Input N
	size_t n = 0;
	printf("[-] Please enter N: ");
	scanf("%uld\n", &n);
	// Allocate memory for array
	DTYPE * a = (DTYPE *)malloc(n * sizeof a);
	// Fill array with numbers 1..n
	fill_array(a, n);
	// Print array
	// print_array(a, n);
	// Compute prefix sum
	prefix_sum(a, n);
	// Print array
	// print_array(a, n);
	// Free allocated memory
	free(a);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    // printf(deviceProp.)
	return EXIT_SUCCESS;
}

void ensure(hipError_t error, const char *msg) {
    if(error != hipSuccess) {
        fprintf(stderr, msg);
        fprintf(stderr, " ---- error code: %s\n", hipGetErrorString(error));
        
        exit(EXIT_FAILURE);
    }
}

__global__ void 
HAS(DTYPE *d_a, DTYPE *tmp, size_t n, int step, int d) {
    DTYPE *a[2] = {d_a, tmp};

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n)
        return;
    a[d][id] = a[1-d][id];
    if (id >= step)
        a[d][id] += a[1-d][id - step];
       
}

void prefix_sum(DTYPE *a, size_t n) {
    size_t size = n * sizeof(DTYPE);
    DTYPE *d_a, *tmp;

    ensure(hipMalloc((void**)&d_a, size), "could not allocate d_a in device.");
    ensure(hipMalloc((void**)&tmp, size), "could not allocate tmp in device.");
    
    ensure(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice), "could not copy to device");

    dim3 block(BLOCK_SIZE, 1, 1);
    dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);

    hipEvent_t start, end;
    ensure(hipEventCreate(&start), "could not create event for start.");
    ensure(hipEventCreate(&end), "could not create event for end.");

    ensure(hipEventRecord(start, NULL), "could not record start.");
    
    int d=1;
    for(int step=1; step<n ; step <<= 1, d = 1-d)
        HAS <<< grid, block >>> (d_a, tmp, n, step, d);
    
    hipDeviceSynchronize();

    ensure(hipEventRecord(end, NULL), "could not record end.");
    ensure(hipEventSynchronize(end), "could not sync.");

    float elapsedtime;
    ensure(hipEventElapsedTime(&elapsedtime, start, end), "could not calc elapsed time.");

    printf("Elapsed time in msec = %f\n", elapsedtime);

    if(d)
        ensure(hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost), "could not copy the result");
    else
        ensure(hipMemcpy(a, tmp, size, hipMemcpyDeviceToHost), "could not copy the result");
    
    ensure(hipFree(d_a), "could not free d_a from device.");
    ensure(hipFree(tmp), "could not free tmp from device.");
}

void print_array(DTYPE *a, size_t n) {
	int i;
	printf("[-] array: ");
	for (i = 0; i < n; ++i) {
		printf(DTYPE_FORMAT, a[i]);
        printf(", ");
	}
	printf("\n");
}

void fill_array(DTYPE *a, size_t n) {
	int i;
	for (i = 0; i < n; ++i) {
		a[i] = i + 1;
	}
}